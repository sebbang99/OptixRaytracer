#include "hip/hip_runtime.h"
/*

 * SPDX-FileCopyrightText: Copyright (c) 2019 - 2024  NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>

#include "helpers.h"
#include "whitted.h"
#include <float.h>
#include <iostream>

extern "C" __global__ void __intersection__cube()
{
    const whitted::HitGroupData* sbt_data = reinterpret_cast<whitted::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::AABBs& cube = sbt_data->geometry_data.getAabb();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float t1, t2, t_near = -FLT_MAX, t_far = FLT_MAX;
    float3 normal = make_float3(0.0f, 0.0f, 0.0f);

    // x-axis
    if (ray_dir.x == 0.0f) {
        if (ray_orig.x < cube.min.x || ray_orig.x > cube.max.x) return;
    }
    else {
        t1 = (cube.min.x - ray_orig.x) / ray_dir.x;
        t2 = (cube.max.x - ray_orig.x) / ray_dir.x;

        if (t1 > t2) {
            float tmp = t1;
            t1 = t2;
            t2 = tmp;
        }
        if (t1 > t_near) {
            t_near = t1;
            normal = make_float3(-1.0f, 0.0f, 0.0f);
        }
        if (t2 < t_far) {
            t_far = t2;
        }
        if (t_near > t_far) return;
        if (t_far < 0.0f || t_near > t_far) return;
    }

    // y-axis
    if (ray_dir.y == 0.0f) {
        if (ray_orig.y < cube.min.y || ray_orig.y > cube.max.y) return;
    }
    else {
        t1 = (cube.min.y - ray_orig.y) / ray_dir.y;
        t2 = (cube.max.y - ray_orig.y) / ray_dir.y;

        if (t1 > t2) {
            float tmp = t1;
            t1 = t2;
            t2 = tmp;
        }
        if (t1 > t_near) {
            t_near = t1;
            normal = make_float3(0.0f, 1.0f, 0.0f);
        }
        if (t2 < t_far) {
            t_far = t2;
        }
        if (t_near > t_far) return;
        if (t_far < 0.0f || t_near > t_far) return;
    }

    // z-axis
    if (ray_dir.z == 0.0f) {
        if (ray_orig.z < cube.min.z || ray_orig.z > cube.max.z) return;
    }
    else {
        t1 = (cube.min.z - ray_orig.z) / ray_dir.z;
        t2 = (cube.max.z - ray_orig.z) / ray_dir.z;

        if (t1 > t2) {
            float tmp = t1;
            t1 = t2;
            t2 = tmp;
        }
        if (t1 > t_near) {
            t_near = t1;
            normal = make_float3(0.0f, 0.0f, -1.0f);
        }
        if (t2 < t_far) {
            t_far = t2;
        }
        if (t_near > t_far) return;
        if (t_far < 0.0f || t_near > t_far) return;
    }

    if (t_near == -FLT_MAX)
        return;
    else
        optixReportIntersection(t_near, 0, float3_as_args(normal));
}

extern "C" __global__ void __intersection__cylinder()
{
    const whitted::HitGroupData* sbt_data = reinterpret_cast<whitted::HitGroupData*>(optixGetSbtDataPointer());
    const GeometryData::Cylinder& cylinder = sbt_data->geometry_data.getCylinder();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir = optixGetWorldRayDirection();
    const float ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    const float3 center = cylinder.center;
    const float radius = cylinder.radius;
    const float height = cylinder.height;

    const float3 moved_orig = ray_orig - center;

    float k2 = 1.0f - ray_dir.y * ray_dir.y;
    float k1 = dot(moved_orig, ray_dir) - moved_orig.y * ray_dir.y;
    float k0 = dot(moved_orig, moved_orig) - moved_orig.y * moved_orig.y - radius * radius;

    float h = k1 * k1 - k2 * k0;
    if (h < 0.0f) return;

    h = sqrtf(h);
    float t = (-k1 - h) / k2;

    float y;
    // Check for intersection with cylinder body
    if (t >= ray_tmin && t <= ray_tmax) {
        y = moved_orig.y + t * ray_dir.y;
        if (y > -height && y < height) {
            float3 normal = (moved_orig + t * ray_dir - make_float3(0.0f, y, 0.0f)) / radius;
            optixReportIntersection(t, 0, float3_as_args(normal));
        }
    }

    // Check for intersection with caps
    t = (((y < 0.0f) ? -height : height) - moved_orig.y) / ray_dir.y;
    if (t >= ray_tmin && t <= ray_tmax) {
        if (abs(k1 + k2 * t) < h) {
            float3 normal = make_float3(0.0f, (y < 0.0f ? -1.0f : 1.0f), 0.0f);
            optixReportIntersection(t, 0, float3_as_args(normal));
        }
    }
}


extern "C" {
__constant__ whitted::LaunchParams params;
}

extern "C" __global__ void __intersection__parallelogram()
{
    const whitted::HitGroupData*       sbt_data = reinterpret_cast<whitted::HitGroupData*>( optixGetSbtDataPointer() );
    const GeometryData::Parallelogram& floor    = sbt_data->geometry_data.getParallelogram();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float3 n  = make_float3( floor.plane );
    float  dt = dot( ray_dir, n );
    float  t  = ( floor.plane.w - dot( n, ray_orig ) ) / dt;
    if( t > ray_tmin && t < ray_tmax )
    {
        float3 p  = ray_orig + ray_dir * t;
        float3 vi = p - floor.anchor;
        float  a1 = dot( floor.v1, vi );
        if( a1 >= 0 && a1 <= 1 )
        {
            float a2 = dot( floor.v2, vi );
            if( a2 >= 0 && a2 <= 1 )
            {
                optixReportIntersection( t, 0, float3_as_args( n ), __float_as_uint( a1 ), __float_as_uint( a2 ) );
            }
        }
    }
}


extern "C" __global__ void __intersection__sphere_shell()
{
    const whitted::HitGroupData*     sbt_data = reinterpret_cast<whitted::HitGroupData*>( optixGetSbtDataPointer() );
    const GeometryData::SphereShell& sphere_shell = sbt_data->geometry_data.getSphereShell();

    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_tmin = optixGetRayTmin(), ray_tmax = optixGetRayTmax();

    float3 O = ray_orig - sphere_shell.center;
    float  l = 1 / length( ray_dir );
    float3 D = ray_dir * l;

    float b = dot( O, D ), sqr_b = b * b;
    float O_dot_O = dot( O, O );
    float radius1 = sphere_shell.radius1, radius2 = sphere_shell.radius2;
    float sqr_radius1 = radius1 * radius1, sqr_radius2 = radius2 * radius2;

    // check if we are outside of outer sphere
    if( O_dot_O > sqr_radius2 + params.scene_epsilon )
    {
        if( O_dot_O - sqr_b < sqr_radius2 - params.scene_epsilon )
        {
            float c    = O_dot_O - sqr_radius2;
            float root = sqr_b - c;
            if( root > 0.0f )
            {
                float  t      = -b - sqrtf( root );
                float3 normal = ( O + t * D ) / radius2;
                optixReportIntersection( t * l, GeometryData::SphereShell::HIT_OUTSIDE_FROM_OUTSIDE, float3_as_args( normal ) );
            }
        }
    }
    // else we are inside of the outer sphere
    else
    {
        float c    = O_dot_O - sqr_radius1;
        float root = b * b - c;
        if( root > 0.0f )
        {
            float t = -b - sqrtf( root );
            // do we hit inner sphere from between spheres?
            if( t * l > ray_tmin && t * l < ray_tmax )
            {
                float3 normal = ( O + t * D ) / ( -radius1 );
                optixReportIntersection( t * l, GeometryData::SphereShell::HIT_INSIDE_FROM_OUTSIDE, float3_as_args( normal ) );
            }
            else
            {
                // do we hit inner sphere from within both spheres?
                t = -b + ( root > 0 ? sqrtf( root ) : 0.f );
                if( t * l > ray_tmin && t * l < ray_tmax )
                {
                    float3 normal = ( O + t * D ) / ( -radius1 );
                    optixReportIntersection( t * l, GeometryData::SphereShell::HIT_INSIDE_FROM_INSIDE, float3_as_args( normal ) );
                }
                else
                {
                    // do we hit outer sphere from between spheres?
                    c             = O_dot_O - sqr_radius2;
                    root          = b * b - c;
                    t             = -b + ( root > 0 ? sqrtf( root ) : 0.f );
                    float3 normal = ( O + t * D ) / radius2;
                    optixReportIntersection( t * l, GeometryData::SphereShell::HIT_OUTSIDE_FROM_INSIDE, float3_as_args( normal ) );
                }
            }
        }
        else
        {
            // do we hit outer sphere from between spheres?
            c             = O_dot_O - sqr_radius2;
            root          = b * b - c;
            float  t      = -b + ( root > 0 ? sqrtf( root ) : 0.f );
            float3 normal = ( O + t * D ) / radius2;
            optixReportIntersection( t * l, GeometryData::SphereShell::HIT_OUTSIDE_FROM_INSIDE, float3_as_args( normal ) );
        }
    }
}
